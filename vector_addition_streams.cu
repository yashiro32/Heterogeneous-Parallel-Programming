#include "hip/hip_runtime.h"
#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < len) {
		out[index] = in1[index] + in2[index];
	}
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;
	
	/* float *h_A;
	float *h_B;
	float *h_C; */
	
	hipStream_t stream0, stream1, stream2, stream3;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	
	float *d_A0, *d_B0, *d_C0; // Device memory for stream 0
	float *d_A1, *d_B1, *d_C1; // Device memory for stream 1
	float *d_A2, *d_B2, *d_C2; // Device memory for stream 2
	float *d_A3, *d_B3, *d_C3; // Device memory for stream 3
	
    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");
	
	int SegSize = inputLength / 4;
	
	/* hipHostAlloc((void**)&h_A, sizeof(float) * inputLength, hipHostMallocDefault);
	hipHostAlloc((void**)&h_B, sizeof(float) * inputLength, hipHostMallocDefault);
	hipHostAlloc((void**)&h_C, sizeof(float) * inputLength, hipHostMallocDefault);
	
	hipMemcpy(h_A, hostInput1, sizeof(float) * inputLength, hipMemcpyHostToHost);
	hipMemcpy(h_B, hostInput2, sizeof(float) * inputLength, hipMemcpyHostToHost);
	hipMemcpy(h_C, hostOutput, sizeof(float) * inputLength, hipMemcpyHostToHost); */
	
	hipMalloc((void**)&d_A0, sizeof(float) * SegSize);
	hipMalloc((void**)&d_B0, sizeof(float) * SegSize);
	hipMalloc((void**)&d_C0, sizeof(float) * SegSize);
	
	hipMalloc((void**)&d_A1, sizeof(float) * SegSize);
	hipMalloc((void**)&d_B1, sizeof(float) * SegSize);
	hipMalloc((void**)&d_C1, sizeof(float) * SegSize);
	
	hipMalloc((void**)&d_A2, sizeof(float) * SegSize);
	hipMalloc((void**)&d_B2, sizeof(float) * SegSize);
	hipMalloc((void**)&d_C2, sizeof(float) * SegSize);
	
	hipMalloc((void**)&d_A3, sizeof(float) * SegSize);
	hipMalloc((void**)&d_B3, sizeof(float) * SegSize);
	hipMalloc((void**)&d_C3, sizeof(float) * SegSize);
	
	dim3 dimBlock(256, 1, 1);
	dim3 dimGrid((SegSize-1)/256+1, 1, 1);
	
	for (int i = 0; i < inputLength; i += SegSize*4) {
		hipMemcpyAsync(d_A0, hostInput1+i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(d_B0, hostInput2+i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
		
		
		
		hipMemcpyAsync(d_A1, hostInput1+i+SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(d_B1, hostInput2+i+SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, stream1);
		
		
		
		hipMemcpyAsync(d_A2, hostInput1+i+SegSize*2, SegSize * sizeof(float), hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(d_B2, hostInput2+i+SegSize*2, SegSize * sizeof(float), hipMemcpyHostToDevice, stream2);
		
		
		
		hipMemcpyAsync(d_A3, hostInput1+i+SegSize*3, SegSize * sizeof(float), hipMemcpyHostToDevice, stream3);
		hipMemcpyAsync(d_B3, hostInput2+i+SegSize*3, SegSize * sizeof(float), hipMemcpyHostToDevice, stream3);
		
		
		
		vecAdd<<<dimGrid, dimBlock, 0, stream0>>>(d_A0, d_B0, d_C0, SegSize);
		vecAdd<<<dimGrid, dimBlock, 0, stream1>>>(d_A1, d_B1, d_C1, SegSize);
		vecAdd<<<dimGrid, dimBlock, 0, stream2>>>(d_A2, d_B2, d_C2, SegSize);
		vecAdd<<<dimGrid, dimBlock, 0, stream3>>>(d_A3, d_B3, d_C3, SegSize);
		
		
		hipMemcpyAsync(hostOutput+i, d_C0, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(hostOutput+i+SegSize, d_C1, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(hostOutput+i+SegSize*2, d_C2, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream2);
		hipMemcpyAsync(hostOutput+i+SegSize*3, d_C3, SegSize * sizeof(float), hipMemcpyDeviceToHost, stream3);
		
	}

	// hipMemcpy(hostOutput, h_C, sizeof(float) * inputLength, hipMemcpyHostToHost);

    wbSolution(args, hostOutput, inputLength);
	
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);
	
	hipFree(d_A0);
	hipFree(d_B0);
	hipFree(d_C0);
	hipFree(d_A1);
	hipFree(d_B1);
	hipFree(d_C1);
	hipFree(d_A2);
	hipFree(d_B2);
	hipFree(d_C2);
	hipFree(d_A3);
	hipFree(d_B3);
	hipFree(d_C3);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
	
	/* free(h_A);
	free(h_B);
	free(h_C); */

    return 0;
}
