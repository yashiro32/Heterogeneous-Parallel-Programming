#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

int const TILE_WIDTH = 16;

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows,
                                     int numAColumns, int numBRows,
                                     int numBColumns, int numCRows,
                                     int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH]; 
	
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;
	
  int Row = by * blockDim.y + ty;
  int Col = bx * blockDim.x + tx;
	
  float Cvalue = 0;
	
  int m = numCRows; // Rows of Matrix C which is equal to the row of Matrix A
  int k = numCColumns; // Columns of Matrix C which is equal to the columns of Matrix B
  int n = numAColumns; // Columns of Matrix A which is equal to rows of Matrix B
	
  // Loop over the A and B tiles required to compute the C element
	for (int t = 0; t < (n-1)/TILE_WIDTH + 1; ++t) {
		// Collaborative loading of A and B tiles into shared memory
		if ((Row < m) && (t*TILE_WIDTH+tx < n)) {
			ds_A[ty][tx] = A[Row*n + t*TILE_WIDTH+tx];
		} else {
			ds_A[ty][tx] = 0.0;
		}
		
		if ((t*TILE_WIDTH+ty < n) && (Col < k)) {
			ds_B[ty][tx] = B[(t*TILE_WIDTH+ty)*k + Col];
		} else {
			ds_B[ty][tx] = 0.0;
		}
		
		__syncthreads();
		
		for (int i = 0; i < TILE_WIDTH; ++i) {
			Cvalue += ds_A[ty][i] * ds_B[i][tx];
		}
		
		__syncthreads();
	}
	
	if (Row < m && Col < k) {
		C[Row*k + Col] = Cvalue;
	}
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA =
      ( float * )wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB =
      ( float * )wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  wbTime_stop(Generic, "Importing data and creating memory on host");
  hostC = (float*)malloc(sizeof(float) * (numCRows * numCColumns));

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**)&deviceA, sizeof(float) * (numARows * numAColumns));
  hipMalloc((void**)&deviceB, sizeof(float) * (numBRows * numBColumns));
  hipMalloc((void**)&deviceC, sizeof(float) * (numCRows * numCColumns));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeof(float) * (numARows * numAColumns), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeof(float) * (numBRows * numBColumns), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((numCColumns-1)/TILE_WIDTH+1, (numCRows-1)/TILE_WIDTH+1, 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeof(float) * (numCRows * numCColumns), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
