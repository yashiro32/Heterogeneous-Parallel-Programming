#include "hip/hip_runtime.h"
// Histogram Equalization

#include    <wb.h>

#define HISTOGRAM_LENGTH 256

#define BLOCK_SIZE 128

//@@ insert code here
	
__device__ unsigned char clamp(unsigned char x, unsigned char start, unsigned char end) {
    return min(max(x, start), end);
}

__global__ void convertFloatToChar(float *in, unsigned char *out, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (index < size) {
		out[index] = (unsigned char) (in[index] * 255);
	}
}

__global__ void RGBToGrayScale(unsigned char *rgb, unsigned char *gray, int size, int width, int height, int channels) {
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	
	int index = row*width + col;
	
	if (row < height && col < width) {
		unsigned char r = rgb[index*channels+0];
	    unsigned char g = rgb[index*channels+1];
	    unsigned char b = rgb[index*channels+2];
	
	    gray[index] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
	}
	
}

__global__ void histogram(unsigned char *buffer, unsigned int *histo, int size) {
	/* int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	__shared__ unsigned int histo_private[256];
	
	if (threadIdx.x < 256) {
		histo_private[threadIdx.x] = 0;
	}
	
	__syncthreads();
	
	// Stride is total number of threads
	int stride = blockDim.x * gridDim.x;
	
	while (i < size) {
		atomicAdd(&(histo_private[buffer[i]]), 1);
		i += stride;
	}
	
	__syncthreads();
	
	if (threadIdx.x < 256) {
		atomicAdd(&histo[threadIdx.x], histo_private[threadIdx.x]);
	} */
	
	
	//  compute histogram with a private version in each block
	__shared__ unsigned int histo_private[HISTOGRAM_LENGTH];
	
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	
	//  index of current pixel
	int index = tx+bx*blockDim.x;
	
	//  set initial values of histogram to zero 
	if (tx < HISTOGRAM_LENGTH) histo_private[tx] = 0;
	
	__syncthreads();
	
	
	int stride = blockDim.x*gridDim.x;
	
	//iterate to add values
	while (index < stride)
	{
		atomicAdd(&(histo_private[buffer[index]]), 1);
		index += stride;
	}
	
	__syncthreads();
	
	//copy private histogram to device histogram
	if(tx<256)
	{
		atomicAdd(&(histo[tx]), histo_private[tx]);
	}
	
}

__global__ void scan(unsigned int * input, unsigned int * output, int len) {
	unsigned int start = 2 * blockIdx.x * blockDim.x;
	unsigned int t = threadIdx.x;
	int i = start + t;
	
	__shared__ unsigned int XY[2*BLOCK_SIZE];
	
	if (start + t < len) {
		XY[t] = input[start + t];
	} else {
		XY[t] = 0.0f;
	}
	
	if (start + blockDim.x + t < len) {
		XY[t + blockDim.x] = input[start + blockDim.x + t];
	} else {
		XY[t + blockDim.x] = 0.0f;
	}
	
	__syncthreads();
	
	for (int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
		int index = (threadIdx.x+1)*stride*2 - 1;
		if (index < 2*BLOCK_SIZE) {
			XY[index] += XY[index-stride];
		}
		
		__syncthreads();
	}
	
	for (int stride = BLOCK_SIZE/2; stride < 0; stride /= 2) {
		__syncthreads();
		
		int index = (threadIdx.x+1)*stride*2 - 1;
		
		if (index + stride < 2*BLOCK_SIZE) {
			XY[index+stride] += XY[index];
		}
	}
	
	__syncthreads();
	
	if (i < len) {
		output[i] = XY[t];
		output[start+blockDim.x+t] = XY[t+blockDim.x];
	}
	
}

__global__ void correct_color(unsigned char *uCharImage, float *cdf, float cdfmin, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (index < size) {
		// unsigned char val = uCharImage[index];
	    // uCharImage[index] = clamp(255*(cdf[val] - cdfmin)/(1 - cdfmin), 0, 255);
		uCharImage[index] = min(max(255*(cdf[uCharImage[index]] - cdfmin)/(1 - cdfmin),0.0),255.0);
	}
}

__global__ void convertCharToFloat(unsigned char* input, float *output, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (index < size) {
		output[index] = (float) (input[index]/255.0);
	}
}


float prob(int x, int width, int height)
{
	return 1.0*x/(width*height);
}


int main(int argc, char ** argv) {
    wbArg_t args;
    int imageWidth;
    int imageHeight;
    int imageChannels;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    const char * inputImageFile;

    //@@ Insert more code here
	float *deviceInputImageData;
	float *deviceOutputImageData;
	
	unsigned char *deviceCastImageData;
	unsigned char *deviceGrayScaleData;
	
	unsigned int *deviceHistogram;
	float *deviceHistoScan;
	
	unsigned int *hostHistogram;
	float *hostHistoScan;
	unsigned char *hostCastImageData;
	

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);

    wbTime_start(Generic, "Importing data and creating memory on host");
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
	wbTime_stop(Generic, "Importing data and creating memory on host");

    //@@ insert code here
	hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);
	
	hipMalloc((void**)&deviceInputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels);
	hipMalloc((void**)&deviceOutputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels);
	
	hipMalloc((void**)&deviceCastImageData, sizeof(unsigned char) * imageWidth * imageHeight * imageChannels);
	hipMalloc((void**)&deviceGrayScaleData, sizeof(unsigned char) * imageWidth * imageHeight);
	hipMalloc((void**)&deviceHistogram, sizeof(unsigned int) * HISTOGRAM_LENGTH);
	hipMalloc((void**)&deviceHistoScan, sizeof(float) * HISTOGRAM_LENGTH);
	
	hipMemcpy(deviceInputImageData, hostInputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels, hipMemcpyHostToDevice);
	
	hipMemset(deviceHistogram, 0, sizeof(unsigned int) * HISTOGRAM_LENGTH);
	hipMemset(deviceHistoScan, 0.0f, sizeof(float) * HISTOGRAM_LENGTH);
	
	int imageDataSize = imageWidth * imageHeight * imageChannels;
	int imageSize = imageWidth * imageHeight;
	
	convertFloatToChar<<<(imageDataSize-1)/1024+1, 1024>>>(deviceInputImageData, deviceCastImageData, imageDataSize);

	dim3 dimBlock(1024, 1024);
	dim3 dimGrid((imageWidth-1)/1024+1, (imageHeight-1)/1024+1);
	
	RGBToGrayScale<<<dimGrid, dimBlock>>>(deviceCastImageData, deviceGrayScaleData, imageSize, imageWidth, imageHeight, imageChannels);
    	
	histogram<<<(imageSize-1)/256+1, 256>>>(deviceGrayScaleData, deviceHistogram, imageSize);
	
	hostHistogram = (unsigned int*) malloc(sizeof(unsigned int) * HISTOGRAM_LENGTH);
	memset(hostHistogram, 0, sizeof(unsigned int) * HISTOGRAM_LENGTH);
	hipMemcpy(hostHistogram, deviceHistogram, sizeof(unsigned int) * HISTOGRAM_LENGTH, hipMemcpyDeviceToHost);
	
	hostHistoScan = (float*) malloc(sizeof(float) * HISTOGRAM_LENGTH);
    memset(hostHistoScan, 0.0f, sizeof(float) * HISTOGRAM_LENGTH);
	
	hostHistoScan[0] = prob(hostHistogram[0], imageWidth, imageHeight);
	for (int i = 1; i < 256; i++) {
		hostHistoScan[i] = hostHistoScan[i-1] + prob(hostHistogram[i], imageWidth, imageHeight);
	}
	
	float cdfmin = hostHistoScan[0];
	for (int i = 1; i < 256; i++) {
		cdfmin = min(cdfmin, hostHistoScan[i]);
	}
	
	
	/* hostCastImageData = (unsigned char*) malloc(sizeof(unsigned char) * imageDataSize);
	hipMemcpy(hostCastImageData, deviceCastImageData, sizeof(unsigned char) * imageDataSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < imageDataSize; i++) {
		unsigned char val = hostCastImageData[i];
		hostCastImageData[i] = clamp(255*(hostHistoScan[val] - cdfmin)/(1 - cdfmin), 0, 255);
	}
	
	for (int i = 0; i < imageDataSize; i++) {
		hostOutputImageData[i] = (float) (hostCastImageData[i]/255.0);
	} */
	
	
	
	hipMemcpy(deviceHistoScan, hostHistoScan, sizeof(float) * HISTOGRAM_LENGTH, hipMemcpyHostToDevice);
	correct_color<<<(imageDataSize-1)/256+1, 256>>>(deviceCastImageData, deviceHistoScan, cdfmin, imageDataSize);
	
	convertCharToFloat<<<(imageDataSize-1)/256+1, 256>>>(deviceCastImageData, deviceOutputImageData, imageDataSize);
	// hipMemcpy(hostOutputImageData, deviceOutputImageData, sizeof(float) * imageDataSize, hipMemcpyDeviceToHost);
	
    wbSolution(args, outputImage);

    //@@ insert code here
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceCastImageData);
	hipFree(deviceGrayScaleData);
	hipFree(deviceHistogram);
	hipFree(deviceHistoScan);
	
	free(hostHistogram);
	free(hostHistoScan);
	// free(hostCastImageData);
	
	wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}

